#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2026, Cuhksz DragonPass. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <algorithm>
#include <cmath>
#include "resize.cuh"

AffineMatrix input_to_infer;
AffineMatrix infer_to_input;

void generate_affine_matrix(int src_width, int src_height, int dst_width, int dst_height) {
    // 获取比例系数
    float scaleIN = std::min(
        static_cast<float>(dst_height) / src_height,
        static_cast<float>(dst_width) / src_width
    );
    float scaleOUT = 1.f / scaleIN;

    // 计算从input到infer仿射变换矩阵各点的值
    input_to_infer.value[0] = scaleIN;
    input_to_infer.value[1] = 0;
    input_to_infer.value[2] = -scaleIN * src_width * 0.5 + dst_width * 0.5;
    input_to_infer.value[3] = 0;
    input_to_infer.value[4] = scaleIN;
    input_to_infer.value[5] = -scaleIN * src_height * 0.5 + dst_height * 0.5;

    // 计算从infer到input仿射变换矩阵各点的值
    infer_to_input.value[0] = scaleOUT;
    infer_to_input.value[1] = 0;
    infer_to_input.value[2] = -scaleOUT * dst_width * 0.5 + src_width * 0.5;
    infer_to_input.value[3] = 0;
    infer_to_input.value[4] = scaleOUT;
    infer_to_input.value[5] = -scaleOUT * dst_height * 0.5 + src_height * 0.5;
}
